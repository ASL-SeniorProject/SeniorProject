
#include "hip/hip_runtime.h"

#include <time.h>

#include <stdio.h>
#include <stdlib.h>
//#include <atlimage.h>


enum color_transform_t
{
	grayscale,
	sRGB,
	LAB
};

enum transform_t
{
	Gaussian
};

#define SIZE 1000

//typedef struct
//{
//	int r;
//	int g;
//	int b;
//} rgb_t;
//
//typedef rgb_t* rgb_ptr;
//typedef rgb_ptr* rgb_list;
//typedef rgb_list* rgb_mat;
//

hipError_t transform(uchar3 *dst_img, uchar3 *src_img, int img_size, int block_size, int grid_size, color_transform_t type);
hipError_t transform();

// convert one scanline to grayscale in parallel
__global__ void grayscale_transform(uchar3 *dst_img, uchar3 *src_img, int img_size)
{
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int idx = y * SIZE + x;

	uchar3 rgb = src_img[idx];
	int average = (rgb.x + rgb.y + rgb.z) / 3;

	dst_img[idx].x = average;
	dst_img[idx].y = average;
	dst_img[idx].z = average;
}

void host_grayscale(uchar3 *dst_img, uchar3 *src_img, int img_size)
{
	for (int i = 0; i < SIZE * SIZE; i++)
	{
		uchar3 rgb = src_img[i];
		int average = (rgb.x + rgb.y + rgb.z) / 3;
		dst_img[i].x = average;
		dst_img[i].y = average;
		dst_img[i].z = average;
	}
}

int main()
{
	// genreate a dummy image
	int size = SIZE * SIZE;
	int img_size = size * sizeof(uchar3);
	int block_size = size / SIZE;
	int grid_size = size / block_size;

	//CImage img;
	uchar3 *src_img, *gray_img, srgb;

	src_img = (uchar3*)malloc(img_size);
	gray_img = (uchar3*)malloc(img_size);

	for (int i = 0; i < SIZE * SIZE; i++)
	{
		uchar3 src, gray;
		
		src.x = 128;
		src.y = 64;
		src.x = 256;

		gray.x = 0;
		gray.y = 0;
		gray.z = 0;

		src_img[i] = src;
		gray_img[i] = gray;
	}

	hipError_t cudaStatus = transform(gray_img, src_img, img_size, block_size, grid_size, grayscale);
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess)
	{
	    fprintf(stderr, "cudadevicereset failed!");
	    return 1;
	}

	clock_t begin = clock();
	host_grayscale(gray_img, src_img, img_size);
	clock_t end = clock();
	double time_spent = 1000 * (double)(end - begin) / CLOCKS_PER_SEC;
	printf("CPU Execution Time: %32fms", time_spent);
	
	free(gray_img);
	free(src_img);
		
	return 0;
	system("pause");
	
	return 0;
	system("pause");
}

// transform an image
hipError_t transform(uchar3 *dst_img, uchar3 *src_img, int img_size, int block_size, int grid_size, color_transform_t type)
{
	hipError_t cudaStatus;
	uchar3 *t_src, *gpu_output;

	cudaStatus = hipMalloc((void**)&t_src, img_size);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMalloc failed!");

	cudaStatus = hipMalloc((void**)&gpu_output, img_size);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMalloc failed!");

	cudaStatus = hipMemcpy(t_src, src_img, img_size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMemcpy failed!");

	float et;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	if (type == grayscale)
		grayscale_transform<<<grid_size, block_size>>>(gpu_output, t_src, img_size);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&et, start, stop);
	printf("GPU Execution Time: %32fms\n", et);
	//// Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);

    //// Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(dst_img, gpu_output, img_size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
        fprintf(stderr, "hipMemcpy failed!");

	return cudaStatus;
}

//cudaError_t transform(rgb_mat dst_img, rgb_mat src_img, unsigned int img_size,  color_transform_t type);
//cudaError_t transform(rgb_t ***image, unsigned int line,  transform_t type);
//
//// convert one scanline to grayscale in parallel
//__global__ void grayscale_transform(rgb_list gpu_output, rgb_mat t_src, unsigned int line)
//{
//	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
//	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
//	unsigned int idx = y * 100 + x;
//
//	rgb_ptr src_rgb = t_src[y][x];
//
//	rgb_ptr gpu_rgb = nullptr;
//	gpu_rgb->r = 0;
//	gpu_rgb->g = 0;
//	gpu_rgb->b = 0;
//
//	double average = (src_rgb->r + src_rgb->g + src_rgb->b) / 3;
//
//	gpu_rgb->r = average;
//	gpu_rgb->g = average;
//	gpu_rgb->b = average;
//
//	gpu_output[idx] = gpu_rgb;
//}
//
//int main()
//{
//	// genreate a dummy image
//	int img_size = 100 * 100 * sizeof(rgb_ptr);
//	int block_size = 4;
//	int grid_size = 100000 / block_size;
//
//	rgb_mat src_img = (rgb_list*)malloc(100 * sizeof(rgb_list));
//	rgb_mat gray_img = (rgb_list*)malloc(100 * sizeof(rgb_list));
//
//	for (int y = 0; y < 100; y++)
//	{
//		src_img[y] = (rgb_list)malloc(100 * sizeof(rgb_ptr));
//		gray_img[y] = (rgb_list)malloc(100 * sizeof(rgb_ptr));
//		for (int x = 0; x < 100; x++)
//		{
//			src_img[y][x] = (rgb_ptr)malloc(sizeof(rgb_ptr));
//			gray_img[y][x] = (rgb_ptr)malloc(sizeof(rgb_ptr));
//		}
//	}
//
//
//	int i, j;
//	for (i = 0; i < 100; i++)
//	{
//		for (j = 0; j < 100; j++)
//		{
//			rgb_ptr s_p = src_img[i][j];
//			s_p->r = 128.0;
//			s_p->g = 76.0;
//			s_p->b = 256.0;
//			src_img[i][j] = s_p;
//
//			rgb_ptr g_p = gray_img[i][j];
//			g_p->r = 0.0;
//			g_p->g = 0.0;
//			g_p->b = 0.0;
//			gray_img[i][j] = g_p;
//		}
//	}
//
//	cudaError_t cudaStatus = transform(gray_img, src_img, img_size, block_size, grid_size, grayscale);
//	if (cudaStatus != cudaSuccess) 
//	{
//	    fprintf(stderr, "addWithCuda failed!");
//	    return 1;
//	}
//
//	//cudaStatus = cudaDeviceReset();
//	//if (cudaStatus != cudaSuccess)
//	//{
//	//    fprintf(stderr, "cudadevicereset failed!");
//	//    return 1;
//	//}
//
//	for (i = 0; i < 100; i++)
//	{
//		for (int j = 0; j < 100; j++)
//		{
//			rgb_ptr rgb = gray_img[i][j];
//			printf("%.3f : %.3f : %.3f\n", rgb->r, rgb->g, rgb->b);
//		}
//	}
//
//	free(gray_img);
//	free(src_img);
//	
//	return 0;
//	system("pause");
//}
//
//// transform an image
//cudaError_t transform(rgb_mat dst_img, rgb_mat src_img, unsigned int img_size, int block_size, int grid_size, color_transform_t type)
//{
//	rgb_mat t_src;
//	rgb_list gpu_output;
//	
//	cudaError_t cudaStatus;
//
//	cudaStatus = cudaMalloc((void**)&t_src, img_size);
//	if (cudaStatus != cudaSuccess)
//		fprintf(stderr, "cudaMalloc failed!");
//
//	cudaStatus = cudaMalloc((void**)&gpu_output, img_size);
//	if (cudaStatus != cudaSuccess)
//		fprintf(stderr, "cudaMalloc failed!");
//
//	cudaStatus = cudaMemcpy(t_src, src_img, img_size, cudaMemcpyHostToDevice);
//	if (cudaStatus != cudaSuccess)
//		fprintf(stderr, "cudaMemcpy failed!");
//
//	if (type == grayscale)
//		grayscale_transform<<<grid_size, block_size>>>(gpu_output, t_src, img_size);
//
//	//// Check for any errors launching the kernel
//    cudaStatus = cudaGetLastError();
//    if (cudaStatus != cudaSuccess)
//        fprintf(stderr, "addKernel launch failed: %s\n", cudaGetErrorString(cudaStatus));
//    
//    // cudaDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = cudaDeviceSynchronize();
//    if (cudaStatus != cudaSuccess)
//        fprintf(stderr, "cudaDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//
//    //// Copy output vector from GPU buffer to host memory.
//    cudaStatus = cudaMemcpy(dst_img, gpu_output, img_size, cudaMemcpyDeviceToHost);
//    if (cudaStatus != cudaSuccess)
//        fprintf(stderr, "cudaMemcpy failed!");
//
//	//cudaStatus = cudaFree(gpu_output);
//	//if (cudaStatus != cudaSuccess)
//	//	fprintf(stderr, "cudaFree failed!");
//
//	return cudaStatus;
//}

/// HERE IS A WORKING EXAMPLE
//cudaError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
//
//__global__ void addKernel(int *c, const int *a, const int *b)
//{
//    int i = threadIdx.x;
//    c[i] = a[i] + b[i];
//}
//
//int main()
//{
//    const int arraySize = 5;
//    const int a[arraySize] = { 1, 2, 3, 4, 5 };
//    const int b[arraySize] = { 10, 20, 30, 40, 50 };
//    int c[arraySize] = { 0 };
//
//    // Add vectors in parallel.
//    cudaError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "addWithCuda failed!");
//        return 1;
//    }
//
//    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//        c[0], c[1], c[2], c[3], c[4]);
//
//    // cudaDeviceReset must be called before exiting in order for profiling and
//    // tracing tools such as Nsight and Visual Profiler to show complete traces.
//    cudaStatus = cudaDeviceReset();
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaDeviceReset failed!");
//        return 1;
//    }
//
//    return 0;
//}
//
//// Helper function for using CUDA to add vectors in parallel.
//cudaError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    cudaError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = cudaSetDevice(0);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = cudaMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = cudaMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = cudaMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = cudaMemcpy(dev_a, a, size * sizeof(int), cudaMemcpyHostToDevice);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = cudaMemcpy(dev_b, b, size * sizeof(int), cudaMemcpyHostToDevice);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = cudaGetLastError();
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", cudaGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // cudaDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = cudaDeviceSynchronize();
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = cudaMemcpy(c, dev_c, size * sizeof(int), cudaMemcpyDeviceToHost);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    cudaFree(dev_c);
//    cudaFree(dev_a);
//    cudaFree(dev_b);
//    
//    return cudaStatus;
//}
