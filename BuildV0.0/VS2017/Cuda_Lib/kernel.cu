#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

enum color_transform_t
{
	grayscale,
	sRGB,
	LAB
};

enum transform_t
{
	Gaussian
};

typedef struct rgb_t
{
	float r;
	float g;
	float b;
};

hipError_t transform(rgb_t *image[100][100], unsigned int line,  color_transform_t type);
hipError_t transform(rgb_t *image[100][100], unsigned int line,  transform_t type);

// convert one scanline to grayscale in parallel
__global__ void grayscale_transform(rgb_t **scanline, unsigned int line)
{
	int j = threadIdx.x;
	rgb_t *gpu_rgb = scanline[j];

	gpu_rgb->r /= 255.0;
	gpu_rgb->g /= 255.0;
	gpu_rgb->b /= 255.0;
}

int main()
{
	// genreate a dummy image
	rgb_t *test_img[100][100];
	int line = 100 * sizeof(rgb_t *);

	int i, j;
	for (i = 0; i < 100; i++)
		for (j = 0; j < 100; j++)
		{
			rgb_t *rgb = test_img[i][j];
			rgb->r = 128.0;
			rgb->g = 76;
			rgb->b = 256.0;
		}

	hipError_t cudaStatus = transform(test_img, line, grayscale);
	if (cudaStatus != hipSuccess) 
	{
	    fprintf(stderr, "addWithCuda failed!");
	    return 1;
	}

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess)
	{
	    fprintf(stderr, "cudadevicereset failed!");
	    return 1;
	}
	
	return 0;
}

// transform an image
hipError_t transform(rgb_t *image[100][100], unsigned int line, color_transform_t type)
{
	int i = threadIdx.y;
	rgb_t **scanline = image[i];
	rgb_t *gpu_scanline;
	rgb_t **gray_scanline;

	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)&gpu_scanline, line);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMalloc failed!");

	cudaStatus = hipMalloc((void**)&gray_scanline, line);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMalloc failed!");

	cudaStatus = hipMemcpy(gpu_scanline, scanline, line, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMemcpy failed!");

	if (type == grayscale)
		grayscale_transform<<<1, line>>>(gpu_scanline, line);

	// Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(gray_scanline, gpu_scanline, line, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
        fprintf(stderr, "hipMemcpy failed!");
}

/// HERE IS A WORKING EXAMPLE
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
//
//__global__ void addKernel(int *c, const int *a, const int *b)
//{
//    int i = threadIdx.x;
//    c[i] = a[i] + b[i];
//}
//
//int main()
//{
//    const int arraySize = 5;
//    const int a[arraySize] = { 1, 2, 3, 4, 5 };
//    const int b[arraySize] = { 10, 20, 30, 40, 50 };
//    int c[arraySize] = { 0 };
//
//    // Add vectors in parallel.
//    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addWithCuda failed!");
//        return 1;
//    }
//
//    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//        c[0], c[1], c[2], c[3], c[4]);
//
//    // hipDeviceReset must be called before exiting in order for profiling and
//    // tracing tools such as Nsight and Visual Profiler to show complete traces.
//    cudaStatus = hipDeviceReset();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceReset failed!");
//        return 1;
//    }
//
//    return 0;
//}
//
//// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    hipError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = hipSetDevice(0);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    
//    return cudaStatus;
//}
