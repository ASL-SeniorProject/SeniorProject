
#include "hip/hip_runtime.h"

#include <time.h>

#include <stdio.h>
#include <stdlib.h>
//#include <atlimage.h>


enum color_transform_t
{
	grayscale,
	sRGB,
	LAB
};

enum transform_t
{
	Gaussian
};

#define SIZE 1000

hipError_t transform(uchar3 *dst_img, uchar3 *src_img, int img_size, int block_size, int grid_size, color_transform_t type);
hipError_t transform();

// convert one scanline to grayscale in parallel
__global__ void grayscale_transform(uchar3 *dst_img, uchar3 *src_img, int img_size)
{
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int idx = y * SIZE + x;

	uchar3 rgb = src_img[idx];
	int average = (rgb.x + rgb.y + rgb.z) / 3;

	dst_img[idx].x = average;
	dst_img[idx].y = average;
	dst_img[idx].z = average;
}

void host_grayscale(uchar3 *dst_img, uchar3 *src_img, int img_size)
{
	for (int i = 0; i < SIZE * SIZE; i++)
	{
		uchar3 rgb = src_img[i];
		int average = (rgb.x + rgb.y + rgb.z) / 3;
		dst_img[i].x = average;
		dst_img[i].y = average;
		dst_img[i].z = average;
	}
}

int main()
{
	// genreate a dummy image
	int size = SIZE * SIZE;
	int img_size = size * sizeof(uchar3);

	// So GPU Programming is somewhat different than regular programming
	// or regular concurrent programming for that matter. 
	// With the GPU, we have to imagine the hardware as such:
	//		The GPU contains 
	int block_size = size / SIZE;
	int grid_size = size / block_size;

	//CImage img;
	uchar3 *src_img, *gray_img, srgb;

	src_img = (uchar3*)malloc(img_size);
	gray_img = (uchar3*)malloc(img_size);

	for (int i = 0; i < SIZE * SIZE; i++)
	{
		uchar3 src, gray;
		
		src.x = 128;
		src.y = 64;
		src.x = 256;

		gray.x = 0;
		gray.y = 0;
		gray.z = 0;

		src_img[i] = src;
		gray_img[i] = gray;
	}

	hipError_t cudaStatus = transform(gray_img, src_img, img_size, block_size, grid_size, grayscale);
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess)
	{
	    fprintf(stderr, "cudadevicereset failed!");
	    return 1;
	}

	clock_t begin = clock();
	host_grayscale(gray_img, src_img, img_size);
	clock_t end = clock();
	double time_spent = 1000 * (double)(end - begin) / CLOCKS_PER_SEC;
	printf("CPU Execution Time: %32fms", time_spent);
	
	free(gray_img);
	free(src_img);
		
	return 0;
	system("pause");
	
	return 0;
	system("pause");
}

// transform an image
hipError_t transform(uchar3 *dst_img, uchar3 *src_img, int img_size, int block_size, int grid_size, color_transform_t type)
{
	hipError_t cudaStatus;
	uchar3 *t_src, *gpu_output;

	cudaStatus = hipMalloc((void**)&t_src, img_size);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMalloc failed!");

	cudaStatus = hipMalloc((void**)&gpu_output, img_size);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMalloc failed!");

	cudaStatus = hipMemcpy(t_src, src_img, img_size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "hipMemcpy failed!");

	float et;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	if (type == grayscale)
		grayscale_transform<<<grid_size, block_size>>>(gpu_output, t_src, img_size);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&et, start, stop);
	printf("GPU Execution Time: %32fms\n", et);
	//// Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);

    //// Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(dst_img, gpu_output, img_size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
        fprintf(stderr, "hipMemcpy failed!");

	return cudaStatus;
}

/// HERE IS A WORKING EXAMPLE
//cudaError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
//
//__global__ void addKernel(int *c, const int *a, const int *b)
//{
//    int i = threadIdx.x;
//    c[i] = a[i] + b[i];
//}
//
//int main()
//{
//    const int arraySize = 5;
//    const int a[arraySize] = { 1, 2, 3, 4, 5 };
//    const int b[arraySize] = { 10, 20, 30, 40, 50 };
//    int c[arraySize] = { 0 };
//
//    // Add vectors in parallel.
//    cudaError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "addWithCuda failed!");
//        return 1;
//    }
//
//    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//        c[0], c[1], c[2], c[3], c[4]);
//
//    // cudaDeviceReset must be called before exiting in order for profiling and
//    // tracing tools such as Nsight and Visual Profiler to show complete traces.
//    cudaStatus = cudaDeviceReset();
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaDeviceReset failed!");
//        return 1;
//    }
//
//    return 0;
//}
//
//// Helper function for using CUDA to add vectors in parallel.
//cudaError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    cudaError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = cudaSetDevice(0);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = cudaMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = cudaMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = cudaMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = cudaMemcpy(dev_a, a, size * sizeof(int), cudaMemcpyHostToDevice);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = cudaMemcpy(dev_b, b, size * sizeof(int), cudaMemcpyHostToDevice);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = cudaGetLastError();
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", cudaGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // cudaDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = cudaDeviceSynchronize();
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = cudaMemcpy(c, dev_c, size * sizeof(int), cudaMemcpyDeviceToHost);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    cudaFree(dev_c);
//    cudaFree(dev_a);
//    cudaFree(dev_b);
//    
//    return cudaStatus;
//}
